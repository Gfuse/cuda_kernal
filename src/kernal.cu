#include <iostream>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void add(int *a, int *b, int *c) {
  *c = *a + *b;
  printf("Hello, world from the device!\n");
}



int main(){

  int a, b, c=0; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Setup input values
  a = 2;
  b = 7;
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  add<<<1,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  std::cout<<c<<'\n';
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;

}
